#include "bench_xgpu.h"

#include <iostream>
#include <cassert>
#include <hip/hip_runtime.h>

#include "xgpu.h"
#include "xgpu_info.h"
#include "util.h"


#define NR_BITS 16
#define NR_CHANNELS 50
#define NR_POLARIZATIONS 2
#define NR_SAMPLES 16
#define NR_RECEIVERS 64
#define NR_BASELINES ((NR_RECEIVERS) * ((NR_RECEIVERS) + 1) / 2)
#define NR_RECEIVERS_PER_BLOCK 32
#define NR_TIMES_PER_BLOCK (128 / (NR_BITS))

#define INPUT_SIZE (NR_RECEIVERS * NR_CHANNELS * NR_SAMPLES * NR_POLARIZATIONS)
#define OUTPUT_SIZE (NR_BASELINES * NR_CHANNELS * NR_POLARIZATIONS * NR_POLARIZATIONS)

typedef struct XGPUInternalContextPartStruct {
  int device;
  std::complex<float> *array_d[2];
  std::complex<float> *matrix_d;
} XGPUInternalContextPart;


inline void checkXGPUCall(int xgpu_error) { 
// void checkXGPUCall(int xgpu_error) { 
    if(xgpu_error != XGPU_OK) {
        std::cerr << __FILE__ << "(" << __LINE__ << ") xGPU error (code " << xgpu_error << ")\n";
        // xgpuFree(xgpu_context);
        exit(1);
    }
}


/* Data ordering for xGPU input vectors is (running from slowest to fastest)
 * [time][channel][station][polarization][complexity]
 * Output matrix has ordering
 * [channel][station][station][polarization][polarization][complexity] (REGISTER TILE TRIANGULAR!!)
 */

/* Define MATRIX_ORDER based on which MATRIX_ORDER_XXX is defined.
 * There are three matrix packing options:
 *
 * TRIANGULAR_ORDER
 * REAL_IMAG_TRIANGULAR_ORDER
 * REGISTER_TILE_TRIANGULAR_ORDER (default)
 *
 * To specify the matrix ordering scheme at library compile time, use one of
 * these options to the compiler:
 *
 * -DMATRIX_ORDER_TRIANGULAR
 * -DMATRIX_ORDER_REAL_IMAG
 * -DMATRIX_ORDER_REGISTER_TILE
 */

/* Return values from xgpuCudaXengine()
#define XGPU_OK                          (0)
#define XGPU_OUT_OF_MEMORY               (1)
#define XGPU_CUDA_ERROR                  (2)
#define XGPU_INSUFFICIENT_TEXTURE_MEMORY (3)
#define XGPU_NOT_INITIALIZED             (4)
#define XGPU_HOST_BUFFER_NOT_SET         (5)
*/

void showxgpuInfo(XGPUInfo xgpu_info) {
    std::cout << "\t=============== XGPU INFO ================\n";
    std::cout << "\tnpol:               " << xgpu_info.npol << "\n";
    std::cout << "\tnstation:           " << xgpu_info.nstation << "\n";
    std::cout << "\tnbaseline:          " << xgpu_info.nbaseline << "\n";
    std::cout << "\tnfrequency:         " << xgpu_info.nfrequency << "\n";
    std::cout << "\tntime:              " << xgpu_info.ntime << "\n";
    std::cout << "\tntimepipe:          " << xgpu_info.ntimepipe << "\n";

    std::cout << "\tcompute_type        ";
    switch(xgpu_info.compute_type) {
        case XGPU_INT8:    std::cout << "INT8 multiply, INT32 accumulate\n"; break;
        case XGPU_FLOAT32: std::cout << "FP32 mulitply,  FP32 accumulate\n"; break;
        default:           std::cout << "<unknown type code: " << xgpu_info.compute_type << ">\n";
    }
    std::cout << "\t=============== EXTRA INFO ===============\n";
    std::cout << "\tinput_type          ";
    switch(xgpu_info.input_type) {
        case XGPU_INT8:    std::cout << "INT8\n"; break;
        case XGPU_INT32:   std::cout << "INT32\n"; break;
        case XGPU_FLOAT32: std::cout << "FP32\n"; break;
        default:           std::cout << "<unknown type code: " << xgpu_info.input_type << ">\n";
    }
    std::cout << "\tvecLength:          " << xgpu_info.vecLength << " (" << byteToMB(xgpu_info.vecLength*sizeof(ComplexInput)) << " MB)\n";
    std::cout << "\tvecLengthPipe:      " << xgpu_info.vecLengthPipe << "\n";
    std::cout << "\tmatLength:          " << xgpu_info.matLength << " (" << byteToMB(xgpu_info.matLength*sizeof(Complex)) << " MB)\n";
    std::cout << "\ttriLength:          " << xgpu_info.triLength << " (" << byteToMB(xgpu_info.triLength*sizeof(Complex)) << " MB)\n";
    std::cout << "\tmatrix_order:       ";
    switch(xgpu_info.matrix_order) {
        case TRIANGULAR_ORDER:               std::cout << "triangular\n"; break;
        case REAL_IMAG_TRIANGULAR_ORDER:     std::cout << "real imaginary triangular\n"; break;
        case REGISTER_TILE_TRIANGULAR_ORDER: std::cout << "register tile triangular\n"; break;
        default: printf("<unknown order code: %d>\n", xgpu_info.matrix_order);
    }

    std::cout << "\tshared_atomic_size: " << xgpu_info.shared_atomic_size << "\n";
    std::cout << "\tcomplex_block_size: " << xgpu_info.complex_block_size << "\n";
}

Results runXGPU(Parameters params, std::complex<float>* samples_h, std::complex<float>* visibilities_h) {
    Results result;
    int device = 0;

    // xGPU has two input buffers, assume this is for read/write ping pong style?
    std::complex<float> *array_d0; // xGPU buffer holding 1st half of input data
    std::complex<float> *array_d1; // xGPU buffer holding 2nd half of input data
    std::complex<float> *matrix_d;

    // allocate GPU X-engine memory
    std::cout << "Initialising XGPU...\n";

    XGPUInfo xgpu_info;
    xgpuInfo(&xgpu_info); // get xGPU info from library
    showxgpuInfo(xgpu_info);

    // check that compiled parameters are equal to the target runtime parameters
    assert(xgpu_info.npol == params.npol &&  "xGPU npol does not match");
    assert(xgpu_info.nstation == params.nstation &&  "xGPU nstation does not match");
    assert(xgpu_info.nfrequency == params.nfrequency && "xGPU nfrequency does not match");
    assert(xgpu_info.ntime == params.nsample &&       "xGPU npol does not match");
    assert(xgpu_info.nbaseline == params.nbaseline && "xGPU npol does not match");
    assert(xgpu_info.vecLength == params.input_size &&   "xGPU vecLength does not match");
    assert(xgpu_info.triLength == params.output_size &&  "xGPU triLength does not match");
    // xgpu_info.matLength will be different because of REGISTER_TILE_TRIANGULAR_ORDER

    XGPUContext xgpu_ctx;
    xgpu_ctx.array_h = NULL; // NOT USED IN MWAX: host input array
    xgpu_ctx.matrix_h = NULL; // USED IN MWAX: results from channel averaging, largely reduced size
    checkXGPUCall(xgpuInit(&xgpu_ctx, device)); // allocates all internal buffers
    XGPUInternalContextPart *xgpuInternalPointer = (XGPUInternalContextPart *)xgpu_ctx.internal;

    // set device pointers equal to buffers created by xGPU
    array_d0 = xgpuInternalPointer->array_d[0]; // location of the 1st xGPU input array, for telling the pre-correlation code where to write results
    array_d1 = xgpuInternalPointer->array_d[1]; // location of the 2nd xGPU input array, for telling the pre-correlation code where to write results
    matrix_d = xgpuInternalPointer->matrix_d;   // the xGPU output matrix, for use in frequency averaging and fetching of visibilities

    hipMemcpy(samples_h, array_d0, params.input_size * sizeof(ComplexInput), hipMemcpyHostToDevice);

    std::cout << xgpuInternalPointer->array_d[0] << std::endl;
    std::cout << array_d0 << std::endl;

    std::cout << xgpuInternalPointer->matrix_d << std::endl;
    std::cout << matrix_d << std::endl;


    result.in_reorder_time = 0;
    result.compute_time = 0;
    result.out_reorder_time = 0;

    checkXGPUCall(xgpuCudaXengine(&(xgpu_ctx), SYNCOP_SYNC_COMPUTE));

    hipMemcpy(matrix_d, visibilities_h, params.output_size * sizeof(ComplexInput), hipMemcpyDeviceToHost);

    xgpuFree(&xgpu_ctx);

    return result;
}
