#include "hip/hip_runtime.h"
#include "bench_mwax_tcc.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <hip/hip_fp16.h>
#include <complex>
#include <chrono>

#include "libtcc/Correlator.h"
#include "util.h"

// parameters required by TCC that are customizable
// NR_BITS can be 4, 8, 16
// only have support for 16 (FP16->FP32) so far
#define NR_BITS 16
#define NR_RECEIVERS_PER_BLOCK 32
#define NR_TIMES_PER_BLOCK (128 / (NR_BITS))

#define checkCudaCall(function, ...) { \
    hipError_t error = function; \
    if (error != hipSuccess) { \
        std::cerr  << __FILE__ << "(" << __LINE__ << ") CUDA ERROR: " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
}

// [station][polarisation][time][frequency] -> [frequency][time / tpb][station][polarisation][tpb]
// need to cast input and output as float, as float-to-half conversion is not supported for complex types
// TODO: change so that number of channels can be > 1024 ... very important for MWAX with 6400 channels
__global__ void mwax_transpose_to_TCC_kernel(Parameters params, const float* input, __half* output) {
    int t, f, s, p;
    f = threadIdx.x;
    t = blockIdx.x;
    p = blockIdx.y;
    s = blockIdx.z;
    
    // split into two time axes
    int t0, t1;
    t0 = t / NR_TIMES_PER_BLOCK;
    t1 = t % NR_TIMES_PER_BLOCK;
    
    int in_idx = 2*(s*params.npol*params.nsample*params.nfrequency + p*params.nsample*params.nfrequency + t*params.nfrequency + f);
    int out_idx = 2*(f*params.nsample*params.nstation*params.npol + t0*params.nstation*params.npol*NR_TIMES_PER_BLOCK 
        + s*params.npol*NR_TIMES_PER_BLOCK + p*NR_TIMES_PER_BLOCK + t1);

    output[out_idx] = __float2half(input[in_idx]);     // real 
    output[out_idx+1] = __float2half(input[in_idx+1]); // complex
}

inline void mwax_transpose_to_TCC(Parameters params, const std::complex<float>* input, std::complex<__half>* output, hipStream_t stream) {
    dim3 block(params.nfrequency, 1, 1);
    dim3 grid(params.nsample, params.npol, params.nstation);

    mwax_transpose_to_TCC_kernel<<<grid, block, 0, stream>>>(params, (float*)input, (__half*)output);
}

void mwax_showTccInfo(Parameters params) {
    std::cout << "\t============== MWAX TCC INFO ==============\n";
    std::cout << "\tnpol:                 " << params.npol << "\n";
    std::cout << "\tnstation:             " << params.nstation << "\n";
    std::cout << "\tnbaseline:            " << params.nbaseline << "\n";
    std::cout << "\tnfrequency:           " << params.nfrequency << "\n";
    std::cout << "\tnsamples:             " << params.nsample << "\n";
    std::cout << "\tcompute_type:         ";
    switch(NR_BITS) { 
        case 4:  std::cout << "INT4 multiply, INT32 accumulate\n"; break;
        case 8:  std::cout << "INT8 multiply, INT32 accumulate\n"; break;
        case 16: std::cout << "FP16 multiply,  FP32 accumulate\n"; break;
    }
    std::cout << "\t=============== EXTRA INFO ===============\n";
    std::cout << "\tinput_size:           " << params.input_size<< " (" << byteToMB(params.input_size*NR_BITS/8*sizeof(half)) << " MB)\n";
    std::cout << "\toutput_size:          " << params.output_size << " (" << byteToMB(params.output_size*NR_BITS/8*sizeof(float)) << " MB)\n";
    std::cout << "\tnreceivers_per_block: " << NR_RECEIVERS_PER_BLOCK << "\n";
    std::cout << "\tntime_per_block:      " << NR_TIMES_PER_BLOCK << "\n";
}

Results runMWAXTCC(Parameters params, const std::complex<float>* input_h, std::complex<float>* visibilities_h) {
    Results result = {0};
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time_ms;

    std::cout << "Initialising & compiling MWAX_TCC kernel with NVRTC...\n";

    checkCudaCall(hipSetDevice(0)); // combine the CUDA runtime API and CUDA driver API
    checkCudaCall(hipFree(0));

    hipStream_t stream;
    std::complex<float>* input_d; // store fp32 input
    std::complex<__half> *tcc_in_d; // typecast down to fp16
    std::complex<float> *tcc_out_d;
    std::complex<float> *tcc_out_h = (std::complex<float>*)malloc(params.output_size * sizeof(std::complex<float>)); 
    try {
        tcc::Correlator correlator(NR_BITS, params.nstation, params.nfrequency, params.nsample, params.npol, NR_RECEIVERS_PER_BLOCK, "MWAX");
        // mwax_showTccInfo(params);
        checkCudaCall(hipStreamCreate(&stream));
        checkCudaCall(hipMalloc(&input_d, params.input_size * sizeof(std::complex<float>)));
        checkCudaCall(hipMalloc(&tcc_in_d, params.input_size * sizeof(std::complex<__half>)));
        checkCudaCall(hipMalloc(&tcc_out_d, params.output_size * sizeof(std::complex<float>)));
        checkCudaCall(hipMemcpy(input_d, input_h, params.input_size * sizeof(std::complex<float>), hipMemcpyHostToDevice));
        hipEventRecord(start);
        mwax_transpose_to_TCC(params, input_d, tcc_in_d, stream);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_ms, start, stop);
        result.in_reorder_time = time_ms / 1000;

        hipEventRecord(start);
        correlator.launchAsync((hipStream_t) stream, (hipDeviceptr_t) tcc_out_d, (hipDeviceptr_t) tcc_in_d);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_ms, start, stop);
        result.compute_time = time_ms / 1000;

        checkCudaCall(hipDeviceSynchronize());

        // reorder from TCC to MWAX format
        checkCudaCall(hipMemcpy(tcc_out_h, tcc_out_d, params.output_size * sizeof(std::complex<float>), hipMemcpyDeviceToHost));
        
        // modified version of TCC that writes directly to triangular order
        result.tri_reorder_time = 0;

        typedef std::chrono::high_resolution_clock Clock;
        auto t0 = Clock::now();
        tri_to_mwax(params, tcc_out_h, visibilities_h);
        auto t1 = Clock::now();
        std::chrono::duration<float> elapsed = t1 - t0;
        result.mwax_time = elapsed.count();

        // Free allocated buffers
        checkCudaCall(hipFree(input_d));
        checkCudaCall(hipFree(tcc_in_d));
        checkCudaCall(hipFree(tcc_out_d));
        free(tcc_out_h);

        checkCudaCall(hipStreamDestroy(stream));
    } catch(std::exception &error) { 
        std::cerr << error.what() << std::endl;
    }

    return result;
}

