#include <iostream>
#include <iomanip>
#include <complex>
#include <random>

#include "bench_tcc.h"
#include "bench_xgpu.h"
#include "util.h"

#define checkCudaCall(function, ...) { \
    hipError_t error = function; \
    if (error != hipSuccess) { \
        std::cerr  << __FILE__ << "(" << __LINE__ << ") CUDA ERROR: " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
}

// fill N complex samples into std::complex<float> array
void createRandomSamples(std::complex<float>* samples, size_t N) {
    std::default_random_engine generator;
    std::normal_distribution<float> distribution(0.0, 5.0);
    for(int i=0; i<N; ++i) { 
        samples[i] = {distribution(generator), distribution(generator)};
    }
}

void createTestVector(std::complex<float>* samples, size_t N) { 
    memset(samples, 0, N * sizeof(std::complex<float>));
    samples[0] = {1, 2};
    samples[1] = {1, 3};
}

void printOutputSnapshot(Parameters params, std::complex<float>* data) {
    int idx = 0;
    std::cout.precision(5);
    for(int b = 0; b < 20; b++) { 
        std::cout << "Baseline: " << b << "\n";
        for(int f = 0; f < 4; f++) {
            std::cout << "ch " << f << " | ";
            for(int p = 0; p < params.npol*params.npol; p++) {
                std::cout << std::fixed << data[idx] << " ";
                idx++;
            }
            std::cout << "\n";
        }
        std::cout << "\n\n";
    }  
}

int main () {
    Parameters params;
    params.npol = 2;
    params.nstation = 64;
    params.nsample = 16;
    params.nfrequency = 50;
    params.nbaseline = (params.nstation * (params.nstation + 1)) / 2;
    params.input_size = params.nstation * params.nsample * params.nfrequency * params.npol;
    params.output_size = params.nbaseline * params.nfrequency * params.npol * params.npol;

    std::cout << "Initialising CUDA...\n";
    hipSetDevice(0); // combine the CUDA runtime API and CUDA driver API
    hipFree(0);

    std::cout << "Generating random complex samples...\n";
    // not very good C++, but we want easy compatibility with C libraries so keep raw pointers
    std::complex<float>* samples_h = new std::complex<float>[params.input_size];
    std::complex<float>* visibilities_h = new std::complex<float>[params.output_size];
    
    // data in [antenna][polarisation][time][channel]
    createRandomSamples(samples_h, params.input_size);
    // createTestVector(samples_h, params.input_size);

    std::cout << "First 10 input samples:\n";
    for(int i = 0; i < 10; ++i) { 
        std::cout << samples_h[i] << "\n";
    }

    Results xgpu_result = runXGPU(params, samples_h, visibilities_h);

    printOutputSnapshot(params, visibilities_h);

    memset(visibilities_h, 0, params.output_size * sizeof(std::complex<float>));
    Results tcc_result = runTCC(params, samples_h, visibilities_h);

    printOutputSnapshot(params, visibilities_h);

    delete samples_h;
    delete visibilities_h;
}

