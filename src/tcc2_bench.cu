#include "hip/hip_runtime.h"
#include "tcc2_bench.h"

#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <hip/hip_fp16.h>
#include <complex>
#include <chrono>

#include "libtcc/Correlator.h"
#include "util.h"

// parameters required by TCC that are customizable
// NR_BITS can be 4, 8, 16
// only have support for 16 (FP16->FP32) so far
#define NR_BITS 16
#define NR_RECEIVERS_PER_BLOCK 32
#define NR_TIMES_PER_BLOCK (128 / (NR_BITS))

#define TRANSPOSE_BLOCK_SIZE 512

#define checkCudaCall(function, ...) { \
    hipError_t error = function; \
    if (error != hipSuccess) { \
        std::cerr  << __FILE__ << "(" << __LINE__ << ") CUDA ERROR: " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
}

namespace tcc2 {

// [station][polarisation][time][frequency] -> [frequency][time / tpb][station][polarisation][tpb]
// need to cast input and output as float, as float-to-half conversion is not supported for complex types
__global__ void transpose_to_TCC_kernel(Parameters params, const float* input, __half* output) {
    int t, f, s, p;
    f = blockIdx.x*blockDim.x + threadIdx.x;
    t = blockIdx.y;
    p = blockIdx.z % params.npol;
    s = blockIdx.z / params.npol;
    
    // split into two time axes
    if(f < params.nfrequency) {
        int t0, t1;
        t0 = t / NR_TIMES_PER_BLOCK;
        t1 = t % NR_TIMES_PER_BLOCK;
        
        int in_idx = 2*(s*params.npol*params.nsample*params.nfrequency + p*params.nsample*params.nfrequency + t*params.nfrequency + f);
        int out_idx = 2*(f*params.nsample*params.nstation*params.npol + t0*params.nstation*params.npol*NR_TIMES_PER_BLOCK 
            + s*params.npol*NR_TIMES_PER_BLOCK + p*NR_TIMES_PER_BLOCK + t1);
        
        output[out_idx] = __float2half(input[in_idx]);     // real 
        output[out_idx+1] = __float2half(input[in_idx+1]); // complex
    }
}

inline void transpose_to_TCC(Parameters params, const std::complex<float>* input, std::complex<__half>* output, hipStream_t stream) {
    // need to support > 1024 channels but still want to keep as minor axis
    if(params.nfrequency <= 1024) {
        dim3 block(params.nfrequency, 1, 1);
        dim3 grid(1, params.nsample, params.nstation*params.npol);
        transpose_to_TCC_kernel<<<grid, block, 0, stream>>>(params, (float*)input, (__half*)output);
    } else {
        dim3 block(TRANSPOSE_BLOCK_SIZE, 1, 1);
        dim3 grid(params.nfrequency / TRANSPOSE_BLOCK_SIZE, params.nsample, params.nstation*params.npol);
        transpose_to_TCC_kernel<<<grid, block, 0, stream>>>(params, (float*)input, (__half*)output);
    }
}

// maps native TCC [frequency][baseline][polarisation][polarisation] to MWAX [baseline][frequency][polarisation][polarisation]
// !!! includes baseline reordering !!!
//
//        TCC      
//         r1
//     +--------   
//     | 0 1 3 6   
//  r2 |   2 4 7   idx = r1*(r1+1)/2 + r2
//     |     5 8   
//     |       9   
//
//        MWAX
//         r1
//     +--------     
//     | 0 1 2 3
//  r2 |   4 5 6   idx = r1 + r2*(N-1) - (r2-1)*r2/2
//     |     7 8 
//     |       9
//
__global__ void tcc_to_mwax_kernel(const Parameters params, const float* input, float *output) {
    int r1 = blockIdx.x;
    int r2 = blockIdx.z;
    int f = blockIdx.y*blockDim.y + threadIdx.y;
    int p = threadIdx.x; // 8

    int tri_idx, mwax_idx;

    if(r2 <= r1 && f < params.nfrequency) {
        tri_idx = f*params.nbaseline*blockDim.x + (r1 * (r1 + 1) / 2 + r2)*blockDim.x + p;
        // tri_idx = (r1 * (r1 + 1) / 2 + r2)*params.nfrequency*blockDim.x + f*blockDim.x + p;
        mwax_idx = (r1 + r2*(params.nstation-1) - (r2-1)*r2 / 2)*params.nfrequency*blockDim.x + f*blockDim.x + p;
        output[mwax_idx] = input[tri_idx];
    }
}

#define FREQ_BLOCK_SIZE 32

// each thread is responsible for mapping a polarisation*polarisation*2 (complex) elements
// will still work if number of polarisations is 1 but will be very inneficient
inline void tcc_to_mwax(const Parameters &params, const std::complex<float>* input, std::complex<float>* output, hipStream_t stream) {
    dim3 dimGrid(params.nstation, (params.nfrequency - 1) / FREQ_BLOCK_SIZE + 1, params.nstation);
    dim3 dimBlock(params.npol*params.npol*2, FREQ_BLOCK_SIZE, 1);
    tcc_to_mwax_kernel<<<dimGrid, dimBlock, 0, stream>>>(params, (float*)input, (float*)output);
}

void showInfo(Parameters params) {
    std::cout << "\t================ TCC INFO ================\n";
    std::cout << "\tnpol:                 " << params.npol << "\n";
    std::cout << "\tnstation:             " << params.nstation << "\n";
    std::cout << "\tnbaseline:            " << params.nbaseline << "\n";
    std::cout << "\tnfrequency:           " << params.nfrequency << "\n";
    std::cout << "\tnsamples:             " << params.nsample << "\n";
    std::cout << "\tcompute_type:         ";
    switch(NR_BITS) { 
        case 4:  std::cout << "INT4 multiply, INT32 accumulate\n"; break;
        case 8:  std::cout << "INT8 multiply, INT32 accumulate\n"; break;
        case 16: std::cout << "FP16 multiply,  FP32 accumulate\n"; break;
    }
    std::cout << "\t=============== EXTRA INFO ===============\n";
    std::cout << "\tinput_size:           " << params.input_size<< " (" << byteToMB(params.input_size*NR_BITS/8*sizeof(half)) << " MB)\n";
    std::cout << "\toutput_size:          " << params.output_size << " (" << byteToMB(params.output_size*NR_BITS/8*sizeof(float)) << " MB)\n";
    std::cout << "\tnreceivers_per_block: " << NR_RECEIVERS_PER_BLOCK << "\n";
    std::cout << "\tntime_per_block:      " << NR_TIMES_PER_BLOCK << "\n";
}

Results run(Parameters params, const std::complex<float>* samples_h, std::complex<float>* visibilities_h) {
    Results result = {0};
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time_ms;

    std::cout << "Initialising & compiling TCC kernel with NVRTC...\n";

    checkCudaCall(hipSetDevice(0)); // combine the CUDA runtime API and CUDA driver API
    checkCudaCall(hipFree(0));

    hipStream_t stream;
    std::complex<float>* input_d; // store fp32 input
    std::complex<__half> *tcc_in_d; // typecast down to fp16
    std::complex<float> *tcc_out_d;
    std::complex<float> *tcc_reordered_d;
    std::complex<float> *tcc_reordered_h = (std::complex<float>*)malloc(params.output_size * sizeof(std::complex<float>)); 

    try {
        tcc::Correlator correlator(NR_BITS, params.nstation, params.nfrequency, params.nsample, params.npol, NR_RECEIVERS_PER_BLOCK);
        // showInfo(params);

        checkCudaCall(hipStreamCreate(&stream));
        checkCudaCall(hipMalloc(&input_d, params.input_size * sizeof(std::complex<float>)));
        checkCudaCall(hipMalloc(&tcc_in_d, params.input_size * sizeof(std::complex<__half>)));
        checkCudaCall(hipMalloc(&tcc_out_d, params.output_size * sizeof(std::complex<float>)));
        checkCudaCall(hipMalloc(&tcc_reordered_d, params.output_size * sizeof(std::complex<float>)));
        checkCudaCall(hipMemcpy(input_d, samples_h, params.input_size * sizeof(std::complex<float>), hipMemcpyHostToDevice));


        hipEventRecord(start);
        transpose_to_TCC(params, input_d, tcc_in_d, stream);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_ms, start, stop);
        result.in_reorder_time = time_ms / 1000;

        hipEventRecord(start);
        correlator.launchAsync((hipStream_t) stream, (hipDeviceptr_t) tcc_out_d, (hipDeviceptr_t) tcc_in_d);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_ms, start, stop);
        result.compute_time = time_ms / 1000;

        checkCudaCall(hipDeviceSynchronize());
        
        hipEventRecord(start);
        tcc_to_mwax(params, tcc_out_d, tcc_reordered_d, stream); // swap baseline and frequency with vanilla TCC
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_ms, start, stop);
        result.mwax_time = time_ms / 1000;

        // don't include transfer in reorder time
        checkCudaCall(hipMemcpy(visibilities_h, tcc_reordered_d, params.output_size * sizeof(std::complex<float>), hipMemcpyDeviceToHost));
        
        // Free allocated buffers
        checkCudaCall(hipFree(input_d));
        checkCudaCall(hipFree(tcc_in_d));
        checkCudaCall(hipFree(tcc_out_d));
        checkCudaCall(hipFree(tcc_reordered_d));

        checkCudaCall(hipStreamDestroy(stream));
    } catch(std::exception &error) { 
        std::cerr << error.what() << std::endl;
    }

    return result;
}

}